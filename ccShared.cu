#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <hiprand.h>

using namespace std;

#include <chrono>
using namespace chrono;

#include "JP/CUDA/JPCuda.h"
using namespace nvcuda;

#include "CONSTANTS.h"

template < typename Fa, typename Fb, typename Fc > __global__ void
MatM(
	const	Fa* a
,	const	Fb* b
,			Fc* c
) {
	__shared__	float	A[ 32 ][ 32 ];
	__shared__	float	B[ 32 ][ 32 ];

	auto n = blockIdx.x * 32;
	auto m = blockIdx.y * 32;

	auto $ = 0.;
	for ( auto k = 0; k < M; k += 32 ) {
		A[ threadIdx.y ][ threadIdx.x ] = a[ ( m + threadIdx.y ) * K + k + threadIdx.x ];
		B[ threadIdx.y ][ threadIdx.x ] = b[ ( k + threadIdx.y ) * N + n + threadIdx.x ];
		__syncthreads();

		for ( auto _ = 0; _ < 32; _++ ) $ += A[ threadIdx.y ][ _ ] * B[ _ ][ threadIdx.x ];
	}
	c[ ( m + threadIdx.y ) * N + n + threadIdx.x ] = $;
}

void
Main() {
	CUDAMemory< half >	a( M * K );
	DummyData( a );

	CUDAMemory< half >	b( K * N );
	DummyData( b );

	CUDAMemory< float >	c( M * N );

auto timer = system_clock::now();

	MatM<<< dim3( M / 32, N / 32 ), dim3( 32, 32 ) >>>( a.$, b.$, c.$ );
	hipDeviceSynchronize();

printf( "%ld ns\n", duration_cast<std::chrono::nanoseconds>( system_clock::now() - timer ).count() );
	c.DtoH();
printf( "%ld ns\n", duration_cast<std::chrono::nanoseconds>( system_clock::now() - timer ).count() );

	a.DtoH();
	b.DtoH();

	for ( auto m = 0; m < M; m++ ) {
		for ( auto n = 0; n < N; n++ ) {
			auto $ = 0.;
			for ( auto k = 0; k < K; k++ ) $ += float( a( m * K + k ) ) * float( b( k * N + n ) );
			auto _ = float( c( m * N + n ) );
			if ( abs( $ - _ ) > 0.05 ) cerr << m << ',' << n << ' ' << $ << ':' << _ << ':' << abs( $ - _ ) << endl;
		}
	}
}

int
main( int argc, char* argv[] ) {
	hipDeviceProp_t _;
	hipGetDeviceProperties( &_, 0 );
	cerr << "maxThreadsPerBlock: " << _.maxThreadsPerBlock << endl;
	Main();
}

