#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <hiprand.h>

using namespace std;

#include <chrono>
using namespace chrono;

#include "JP/CUDA/JPCuda.h"
using namespace nvcuda;

template < typename Fa, typename Fb, typename Fc > __global__ void
matmal_16x16(
	const	Fa* a
,	const	Fb* b
,			Fc* c
,	size_t	M
,	size_t	N
,	size_t	K
) {
	auto $ = 0.;
	auto n = blockIdx.x * blockDim.x + threadIdx.x;
	auto m = blockIdx.y * blockDim.y + threadIdx.y;
	for ( auto k = 0; k < K; k++ ) $ += float( a[ m * K + k ] ) * float( b[ k * N + n ] );
	c[ m * N + n ] = $;
}

#define	D	4096
#define	M	D
#define	K	D
#define	N	D

struct
CudaTimer {
	hipEvent_t	start;
	hipEvent_t	stop;

	CudaTimer() {
		_C( hipEventCreate( &start ) );
		_C( hipEventCreate( &stop ) );
		_C( hipEventRecord( start ) );
	}
	~
	CudaTimer() {
		_C( hipEventRecord( stop ) );
		_C( hipEventSynchronize( stop ) );
		float _ = 0;
		_C( hipEventElapsedTime( &_, start, stop ) );
		printf( "CudaTimer: %f ms\n", _ );
		_C( hipEventDestroy( start ) );
		_C( hipEventDestroy( stop ) );
	}
};

void
Main() {

//	cerr << fixed << setprecision( 3 );

	CUDAMemory< half > a( M * K );
	DummyData( a );
	a.DtoH();
//	a.Dump< K >();

	CUDAMemory< half > b( K * N );
	DummyData( b );
	b.DtoH();
//	b.Dump< N >();

	CUDAMemory< float > c( M * N );
//	c.Zeroset();
//	hipDeviceSynchronize();

	auto timer = system_clock::now();
	matmal_16x16<<< dim3( M / 32, N / 32 ), dim3( 32, 32 ) >>>( a.$, b.$, c.$, M, N, K );	//	32: FIXED NUMBER warp size
	//	Managed でやるときはこれが必須！
	hipDeviceSynchronize();
	printf( "%f ms\n", duration_cast<std::chrono::nanoseconds>( system_clock::now() - timer ).count() / 1000000. );
	c.DtoH();
	printf( "%f ms\n", duration_cast<std::chrono::nanoseconds>( system_clock::now() - timer ).count() / 1000000. );
//	c.Dump< N >();

//	for ( auto m = 0; m < M; m++ ) {
//		for ( auto n = 0; n < N; n++ ) {
//			auto $ = 0.;
//			for ( auto k = 0; k < K; k++ ) $ += float( a( m * K + k ) ) * float( b( k * N + n ) );
//			auto _ = float( c( m * N + n ) );
//			if ( abs( $ - _ ) > 0.01 ) cerr << m << ',' << n << ' ' << $ << ':' << _ << ':' << abs( $ - _ ) << endl;
//		}
//	}
}

int
main( int argc, char* argv[] ) {
	hipDeviceProp_t _;
	hipGetDeviceProperties( &_, 0 );
	cerr << "maxThreadsPerBlock: " << _.maxThreadsPerBlock << endl;
	Main();
}

