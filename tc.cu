#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <hiprand.h>

using namespace std;

#include <chrono>
using namespace chrono;

#include "JP/CUDA/JPCuda.h"
using namespace nvcuda;

#include "CONSTANTS.h"

template < typename F > __global__ void
MatPro(
	const	half*	_a
,	const	half*	_b
,			F*		_c
) {
	wmma::fragment< wmma::matrix_a, 16, 16, 16, half, wmma::row_major > a;
	wmma::fragment< wmma::matrix_b, 16, 16, 16, half, wmma::row_major > b;
	wmma::fragment< wmma::accumulator, 16, 16, 16, F > c;

	wmma::fill_fragment( c, 0 );

	for ( auto k = 0; k < K; k += 16 ) {
		wmma::load_matrix_sync( a, _a + ( blockIdx.y * K * 16 + k ), K );
		wmma::load_matrix_sync( b, _b + ( k * N + blockIdx.x * 16 ), N );
		wmma::mma_sync( c, a, b, c );
	}

	wmma::store_matrix_sync( _c + ( blockIdx.y * N * 16 + blockIdx.x * 16 ), c, N, wmma::mem_row_major );
}

void
Main() {

	CUDAMemory< half > a( M * K );
	DummyData( a );

	CUDAMemory< half > b( K * N );
	DummyData( b );

	CUDAMemory< float > c( M * N );

	auto timer = system_clock::now();
	MatPro<<< dim3( N / 16, M / 16 ), 32 >>>( a.$, b.$, c.$ );	//	32: WARP SIZE ( FIXED NUMBER ) 
	hipDeviceSynchronize();
	c.DtoH();
	printf( "%ld ns\n", duration_cast<std::chrono::nanoseconds>( system_clock::now() - timer ).count() );

	a.DtoH();
	b.DtoH();

	c.Host()[ M * N - 1 ] = -1;
	for ( auto m = 0; m < M; m++ ) {
		for ( auto n = 0; n < N; n++ ) {
			auto $ = 0.;
			for ( auto k = 0; k < K; k++ ) {
				$ += float( a.Host()[ m * K + k ] ) * float( b.Host()[ k * N + n ] );
			}
			auto _ = float( c.Host()[ m * N + n ] );
			if ( abs( $ - _ ) > 1 ) {
				cerr << m << ',' << n << ' ' << $ << ':' << _ << ':' << abs( $ - _ ) << endl;
				throw "eh?";
			}
		}
	}
}

int
main( int argc, char* argv[] ) {
	hipDeviceProp_t _;
	hipGetDeviceProperties( &_, 0 );
	cerr << "maxThreadsPerBlock: " << _.maxThreadsPerBlock << endl;
	Main();
}

